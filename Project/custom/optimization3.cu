#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"
#define TILE_WIDTH 16
#define BLOCK_SIZE 1024




__global__ void unroll_and_mul_3(const float* input, float* output, const float* mask, const int Map_out, const int Channel, const int Height, const int Width, const int K, const int Height_out, const int Width_out, const int aCol, const int bCol) {
#define o4(i3, i2, i1, i0) output[(i3) * (Map_out * bCol) + (i2) * bCol + (i1) * Width_out + i0]
#define i4(i3, i2, i1, i0) input[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]
#define mask_4d(i3, i2, i1, i0) mask[(i3) * aCol + (i2) * (K * K) + (i1) * (K) + i0]
	__shared__ float Tile_mask[TILE_WIDTH][TILE_WIDTH];
	__shared__ float Tile_input[TILE_WIDTH][TILE_WIDTH];
        
        int Row = blockIdx.y * TILE_WIDTH + threadIdx.y;
        int Col = blockIdx.x * TILE_WIDTH + threadIdx.x;

	int cnt = ceil((float)aCol / TILE_WIDTH);
	float Pvalue = 0.0;

	int c, h_out, w_out, p, q, col, row;
        for(int i = 0; i < cnt; ++i) {
		col = i * TILE_WIDTH + threadIdx.x;
		row = i * TILE_WIDTH + threadIdx.y;
		c = col / (K * K);
		h_out = (col % (K * K)) / K;
		w_out = (col % (K * K)) % K;

		if(col < aCol && Row < Map_out) {
			Tile_mask[threadIdx.y][threadIdx.x] = mask_4d(Row, c, h_out, w_out);
		}
		else {
			Tile_mask[threadIdx.y][threadIdx.x] = 0.0;
		}

		c = row / (K * K);
		h_out = Col / Width_out;
		w_out = Col % Width_out;

		p = row % (K * K) / K;
		q = (row % (K * K)) % K;

		if(row < aCol && Col < bCol){
			Tile_input[threadIdx.y][threadIdx.x] = i4(blockIdx.z, c, h_out + p, w_out + q);
		}
		else {
			Tile_input[threadIdx.y][threadIdx.x] = 0.0;
		}

		__syncthreads();

		if(Row < Map_out && Col < bCol){
			for(int j = 0; j < TILE_WIDTH; j++) {
				Pvalue += Tile_mask[threadIdx.y][j] * Tile_input[j][threadIdx.x];
			}
		}
		__syncthreads();
	}
	if(Row < Map_out && Col < bCol) {
		o4(blockIdx.z, Row, Col / Width_out, Col % Width_out) = Pvalue;
	}
	
}



__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }

    hipMalloc((void **)device_input_ptr, Batch * Channel * Width * Height * sizeof(float));
    hipMalloc((void **)device_output_ptr, Batch * Map_out * (Width - K + 1) * (Height - K + 1) * sizeof(float));
    hipMalloc((void **)device_mask_ptr, Map_out * Channel * K * K * sizeof(float));

    hipMemcpy(*device_input_ptr, host_input, Batch * Channel * Width * Height * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*device_mask_ptr, host_mask, Map_out * Channel * K * K * sizeof(float), hipMemcpyHostToDevice);
	

}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Set the kernel dimensions and call the kernel
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;

    //const int unroll_Height = Channel * K * K;
    //const int unroll_Width = Height_out * Width_out;

    


    const int aCol = Channel * K * K;
    //const int bRow = Channel * K * K;
    const int bCol = (Height - K + 1) * (Width - K + 1);
    //const int cCol = bCol;


    dim3 DimGrid(ceil((1.0 * Height_out * Width_out) / TILE_WIDTH), ceil((1.0 * Map_out)/TILE_WIDTH), Batch);
    dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1);


    unroll_and_mul_3<<<DimGrid, DimBlock>>>(device_input, device_output, device_mask, Map_out, Channel, Height, Width, K, Height_out, Width_out, aCol, bCol);


    //hipFree(unrolled);



}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Copy the output back to host

    hipMemcpy(host_output, device_output, Batch * Map_out * (Width - K + 1) * (Height - K + 1) * sizeof(float), hipMemcpyDeviceToHost);


    // Free device memory
    hipFree(device_input);
    hipFree(device_output);
    hipFree(device_mask);

}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
